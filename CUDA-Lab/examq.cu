#include "hip/hip_runtime.h"
float* d_A = NULL;      // Pointer to matrix A in device memory
float* h_A = NULL;      // Pointer to matrix A in host memory
int N = 1024;           // X and y dimesions of matrix A
int BLOCK_SIZE = 32;    // X and y dimesions of a thread block

void inc_cpu(float* A, int n) {
        for (int i=0; i<n; i++) {
                for (int j=0; j<n; j++) {
                        A[i*n + j] += 1;
                }
        }
}

__global__ void inc_gpu(float* A, int n)
{
        // 1. TODO: Implement
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n) {
		for (int j = 0; j < n; j++)
			A[i*n + j] += 1
	}
}

int main(int argc, char** argv)
{
        int policy = atoi(argv[1]);             // 1 or 2

        float* h_A = (float*)malloc(sizeof(float) * N * N);
        memset(h_A, 0, sizeof(float) * N * N);

        if(policy==1){ // CPU version
                inc_cpu(h_A, N);
        }
        else if(policy==2){ // GPU version
                // 2. TODO: Set up GPU memory and copy input data
				hipMalloc((void**) &d_A, (n * n * sizeof(float)));
				hipMemcpy(d_A, h_A, sizeof(float) * n * n, hipMemcpyHostToDevice);

                // 3. TODO: Launch kernel inc_gpu
			
        dim2 dimBlock(block_size, block_size);
        dim2 dimGrid(N / dimBlock.x, N / dimBlock.y);
        mm_gpu <<<dimGrid, dimBlock, sizeof(float) * block_size * block_size * 2>>> (d_A, N);

                hipDeviceSynchronize();

                // 4. TODO: Copy back result data and tear down GPU memory
				hipMemcpy(h_A, d_A, (sizeof(float) * n * n), hipMemcpyDeviceToHost);
    			hipFree(d_A);
        }

        free(h_A);
        return 0;
}